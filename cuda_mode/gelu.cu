#include "hip/hip_runtime.h"
/**
 python -m compile_cuda_kernel \
 --name=gelu \
 --cu_file=gelu.cu \
 --cpp_source="torch::Tensor gelu(const torch::Tensor& input);" \
 --funcs=gelu
*/
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA Tensor.")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous.")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}

__global__
void gelu_kernel(float* input, float* output, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float x = input[i];
    output[i] = 0.5f * x * (1.0f + tanhf(sqrtf(2.0f / 3.1415926535897932f) * (x + 0.044715f * x*x*x)));
}

torch::Tensor gelu(const torch::Tensor& input) {
    CHECK_INPUT(input);

    auto output = torch::empty_like(input);

    int threads_per_block = 256;
    int n = input.numel();
    int num_blocks = cdiv(n, threads_per_block);
    gelu_kernel<<<num_blocks, threads_per_block>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        n);

    C10_CUDA_KERNEL_LAUNCH_CHECK();
    return output;
}